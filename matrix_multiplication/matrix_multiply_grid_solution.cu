#include "hip/hip_runtime.h"
  /**
   Square matrix multiplication example

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
           Daniel Campora (dcampora@cern.ch)
   date: 05/2019, 06/2021

 */

#include <chrono>
#include <iostream>
#include <cstdio>

#include "../helpers/helpers.h"
#include "matrix_utils.h"

/**
 * @brief Multiplies matrices using multiple blocks and threads.
 * @details The version shown here is tied to be invoked with enough
 *          blocks and threads (otherwise the results would be wrong).
 *          
 *          A grid and block dimension-strided version is included as an optional
 *          commented out version. This version would work irrespective of
 *          the number of blocks and threads it is invoked with.
 */
__global__ void multiply_square_matrices(const int size,
                                         const float *A,
                                         const float *B,
                                         float *C) {
  // Deal with element i, j
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // Check i and j are in bounds
  if (i < size && j < size) {
    float element = 0;
    for (int k = 0; k < size; k++) {
      element += A[i * size + k] * B[k * size + j];
    }
    C[i * size + j] = element;
  }

  // Optional solution: Grid and block dimension-strided for-loop which works for any grid and block dimension
  // for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
  //   for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < size; j += gridDim.y * blockDim.y) {
  //     float element = 0;
  //     for (int k = 0; k < size; k++) {
  //       element += A[i * size + k] * B[k * size + j];
  //     }
  //     C[i * size + j] = element;
  //   }
  // }
}

int main(int argc, char *argv[]) {

  if (argc != 2) {
    std::cout << "Needs an argument: number of rows (= number of columns) of "
                 "square matrices\n";
    return -1;
  }

  const int matrix_size = atoi(argv[argc - 1]);

  // Allocate host and device memory for three matrices
  float *host_matrix[3]; // matrix[0] and matrix[1] are the source for the
                         // multiplication, result stored in matrix[2]
  float *device_matrix[3];

  for (int i = 0; i < 3; i++) {
    host_matrix[i] = new float[matrix_size * matrix_size];
    CUDA_ASSERT(hipMalloc((void **)&device_matrix[i],
                           matrix_size * matrix_size * sizeof(float)));
  }

  // Initialize matrices
  for (int i = 0; i < matrix_size; i++) {
    for (int j = 0; j < matrix_size; j++) {
      host_matrix[0][i * matrix_size + j] = (i * (j + 1)) % 10;
      host_matrix[1][i * matrix_size + j] = (2 * i + j) % 10;
      host_matrix[2][i * matrix_size + j] = 0;
    }
  }

  // Copy matrices to device
  for (int i = 0; i < 3; i++) {
    CUDA_ASSERT(hipMemcpy(device_matrix[i], host_matrix[i],
                           matrix_size * matrix_size * sizeof(float),
                           hipMemcpyHostToDevice));
  }

  // Launch kernel
  int size = matrix_size;
  int number_of_threads = 32;
  int number_of_blocks = (size + number_of_threads - 1) / number_of_threads;

  dim3 grid(number_of_blocks, number_of_blocks);
  dim3 block(number_of_threads, number_of_threads);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  multiply_square_matrices<<<grid, block>>>(size, device_matrix[0],
                                            device_matrix[1], device_matrix[2]);

  hipDeviceSynchronize();

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  // Copy back result
  CUDA_ASSERT(hipMemcpy(host_matrix[2], device_matrix[2],
                         matrix_size * matrix_size * sizeof(float),
                         hipMemcpyDeviceToHost));

  // Check and print result
  check_result(host_matrix[0], host_matrix[1], host_matrix[2], matrix_size,
               matrix_size, matrix_size);

  std::cout << "Kernel duration: " << elapsed_seconds.count() << " s\n";

  // Free memory
  for (int i = 0; i < 3; i++) {
    delete[] host_matrix[i];
    CUDA_ASSERT(hipFree(device_matrix[i]));
  }
}
