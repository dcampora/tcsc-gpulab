#include "hip/hip_runtime.h"
/**
   Square matrix multiplication example

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
           Daniel Campora (dcampora@cern.ch)
   date: 05/2019, 06/2021
 */

#include "matrix_utils.h"
#include <chrono>
#include <cstdio>
#include <iostream>

/**
 * @brief Multiplication of square matrices without any parallelization.
 * @details In this sequential implementation of square matrix multiplication,
 *          every thread would work on all the elements.
 */
__global__ void multiply_square_matrices(const int size, const float *A,
                                         const float *B, float *C) {
  for (int i = 0; i < size; ++i) {
    for (int j = 0; j < size; ++j) {
      float element = 0;
      for (int k = 0; k < size; k++) {
        element += A[i * size + k] * B[k * size + j];
      }
      C[i * size + j] = element;
    }
  }
}

int main(int argc, char *argv[]) {

  if (argc != 2) {
    std::cout << "Needs an argument: number of rows (= number of columns) of "
                 "square matrices\n";
    return -1;
  }

  const int matrix_size = atoi(argv[argc - 1]);

  // Allocate host and device memory for three matrices
  float *host_matrix[3]; // matrix[0] and matrix[1] are the source for the
                         // multiplication, result stored in matrix[2]
  float *device_matrix[3];

  for (int i = 0; i < 3; i++) {
    host_matrix[i] = new float[matrix_size * matrix_size];
    hipMalloc((void **)&device_matrix[i],
               matrix_size * matrix_size * sizeof(float));
  }

  // Initialize matrices
  for (int i = 0; i < matrix_size; i++) {
    for (int j = 0; j < matrix_size; j++) {
      host_matrix[0][i * matrix_size + j] = (i * (j + 1)) % 10;
      host_matrix[1][i * matrix_size + j] = (2 * i + j) % 10;
      host_matrix[2][i * matrix_size + j] = 0;
    }
  }

  // Copy matrices to device
  for (int i = 0; i < 3; i++) {
    hipMemcpy(device_matrix[i], host_matrix[i],
               matrix_size * matrix_size * sizeof(float),
               hipMemcpyHostToDevice);
  }

  // Launch kernel
  int size = matrix_size;
  dim3 grid(1);
  dim3 block(1);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  multiply_square_matrices<<<grid, block>>>(size, device_matrix[0],
                                            device_matrix[1], device_matrix[2]);

  hipDeviceSynchronize();

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  // Copy back result
  hipMemcpy(host_matrix[2], device_matrix[2],
             matrix_size * matrix_size * sizeof(float), hipMemcpyDeviceToHost);

  // Check and print result
  check_result(host_matrix[0], host_matrix[1], host_matrix[2], matrix_size,
               matrix_size, matrix_size);

  std::cout << "Kernel duration: " << elapsed_seconds.count() << " s\n";

  // Free memory
  for (int i = 0; i < 3; i++) {
    delete[] host_matrix[i];
    hipFree(device_matrix[i]);
  }
}
