#include "hip/hip_runtime.h"
/**
 Square matrix multiplication example

 author: Daniel Campora (dcampora@nvidia.com)
 date: 03/2024

*/

#include <chrono>
#include <cstdio>
#include <iostream>
#include "matrix_utils.h"
#include <mma.h>
using namespace nvcuda;

using storage_T = half;
using arithmetic_T = float;

// Define the tile size
constexpr int TILE_SIZE = 16;

/**
 * @brief Multiplies matrices using Tensor Cores.
 * @details Uses Tensor Cores to perform matrix-matrix multiplication.
 *          Each warp should work on a separate TILE_SIZE * TILE_SIZE fragment.
 */
__global__ void multiply_square_matrices(const int size, const storage_T *A,
                                         const storage_T *B, arithmetic_T *C) {
  // Your solution goes here...
}

int main(int argc, char *argv[]) {
  
  if (argc != 2) {
    std::cout << "Needs an argument: number of rows (= number of columns) of "
                 "square matrices\n";
    return -1;
  }

  const int matrix_size = atoi(argv[argc - 1]);
  
  // Allocate host and device memory for three matrices
  storage_T *host_matrix[2];            // matrix[0] and matrix[1] are the source for the
  arithmetic_T *host_result_matrix; // multiplication, result stored in host_result_matrix
  storage_T *device_matrix[2];
  arithmetic_T *device_result_matrix;

  for (int i = 0; i < 2; i++) {
    host_matrix[i] = new storage_T[matrix_size * matrix_size];
    hipMalloc((void **)&device_matrix[i],
               matrix_size * matrix_size * sizeof(storage_T));
  }
  host_result_matrix = new arithmetic_T[matrix_size * matrix_size];
  hipMalloc((void **)&device_result_matrix,
             matrix_size * matrix_size * sizeof(arithmetic_T));

  // Initialize matrices
  for (int i = 0; i < matrix_size; i++) {
    for (int j = 0; j < matrix_size; j++) {
      host_matrix[0][i * matrix_size + j] = 0.1 * (((i + 1) * (j + 1)) % 10);
      host_matrix[1][i * matrix_size + j] = 0.1 * ((2 * i + j) % 10);
      host_result_matrix[i * matrix_size + j] = 0;
    }
  }
     
  // Copy input matrices to device
  for (int i = 0; i < 2; i++) {
    hipMemcpy(device_matrix[i], host_matrix[i],
               matrix_size * matrix_size * sizeof(storage_T),
               hipMemcpyHostToDevice);
  }
  hipMemcpy(device_result_matrix, host_result_matrix,
             matrix_size * matrix_size * sizeof(arithmetic_T),
             hipMemcpyHostToDevice);

  // Launch kernel
  int size = matrix_size;
  int number_of_threads_y = 4;
  int number_of_threads_z = 4;
  int number_of_blocks = size / (TILE_SIZE * number_of_threads_y * number_of_threads_z);

  dim3 grid(number_of_blocks, number_of_blocks);
  dim3 block(32, number_of_threads_y, number_of_threads_z);

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  multiply_square_matrices<<<grid, block>>>(size, device_matrix[0],
                                            device_matrix[1], device_result_matrix);

  hipDeviceSynchronize();

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  // Copy back result
  hipMemcpy(host_result_matrix, device_result_matrix,
             matrix_size * matrix_size * sizeof(arithmetic_T), hipMemcpyDeviceToHost);

  // Check and print result
  double threshold = 0.1;
  std::vector<double> host_matrix_A_d(matrix_size * matrix_size);
  std::vector<double> host_matrix_B_d(matrix_size * matrix_size);
  for (int i = 0; i < matrix_size; i++) {
    for (int j = 0; j < matrix_size; j++) {
      host_matrix_A_d[i * matrix_size + j] = 0.1 * (((i + 1) * (j + 1)) % 10);
      host_matrix_B_d[i * matrix_size + j] = 0.1 * ((2 * i + j) % 10);
    }
  }
  check_result<double>(host_matrix_A_d.data(), host_matrix_B_d.data(), host_result_matrix, matrix_size,
                       matrix_size, matrix_size, threshold);

  std::cout << "Kernel duration: " << elapsed_seconds.count() << " s\n";

  // Free memory
  for (int i = 0; i < 2; i++) {
    delete[] host_matrix[i];
    hipFree(device_matrix[i]);
  }
  delete[] host_result_matrix;
  hipFree(device_result_matrix);
}
