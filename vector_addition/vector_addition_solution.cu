/**
   Vector addition:
   takes vectors a and b as input, computes vector sum 
   and stores output in vector c

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
   date: 05/2019
   updated: 06/2021

 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

void init_with(int* a, float val, int N) {
    for ( int i = 0; i < N; i++ ) {
        a[i] = val;
    }
}

void vector_addition_cpu(int *a, int *b, int *c, int N) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

void check_elements(int correct, int* vec, int N) {
    for (int i = 0; i < N; i++) {
         if (vec[i] != correct) {
                 printf("ERROR: vec[%u] = %d, should be %d \n", i, vec[i], correct);
         }
    }
}

__global__ void vector_addition_gpu(int *a, int *b, int *c, int size) {
  const int start = threadIdx.x + blockIdx.x * blockDim.x;
  const int stride = blockDim.x * gridDim.x;
  for (int i = start; i < size; i += stride) {
    c[i] = a[i] + b[i];
  }
}

int main(int argc, char *argv[] ) {

  if ( argc != 4 ) {
    cout << "Need three arguments: size of vector, number of threads / block and number of blocks in the grid" << endl;
    return -1;
  }
  
  const int size  = atoi(argv[argc-3]);
  const int n_threads = atoi(argv[argc-2]);
  const int n_blocks = atoi(argv[argc-1]);

  cout << "Adding vectors of size " <<  size << " with " << n_threads << " threads" << " and " << n_blocks << " blocks" << endl;  
 
  /* Host memory for the two input vectors a and b and the output vector c */
  int *a_h = new int[size];
  int *b_h = new int[size];
  int *c_h = new int[size];

  /* Initialize vectors */
  init_with(a_h, 13, size);
  init_with(b_h, 9, size);
  init_with(c_h, 0, size);
  
  /* Device pointers for the three vectors a, b, c */
  int *a_d, *b_d, *c_d;
  hipMalloc( (void**)&a_d, size * sizeof(int) );
  hipMalloc( (void**)&b_d, size * sizeof(int) );
  hipMalloc( (void**)&c_d, size * sizeof(int) );

  /* Copy vectors to device */
  hipMemcpy( a_d, a_h, size * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( b_d, b_h, size * sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( c_d, c_h, size * sizeof(int), hipMemcpyHostToDevice );
  
  /* Define grid dimensions */
  dim3 grid_dim(n_blocks);
  dim3 block_dim(n_threads);

  /* Call kernel */
  vector_addition_gpu<<<grid_dim, block_dim>>>( a_d, b_d, c_d, size);

  hipMemcpy( c_h, c_d, size * sizeof(int), hipMemcpyDeviceToHost );

  /* Make sure GPU work is done */
  hipDeviceSynchronize();

  check_elements(22, c_h, size);

  hipFree( a_d );
  hipFree( b_d );
  hipFree( c_d );

  /* free host memory */
  delete [] a_h;
  delete [] b_h;
  delete [] c_h;

  
  return 0;
}
