/**
   Run Kalman filter on hits belonging to tracks of LHCb's Velo detector

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
   date: 05/2019

 */

#include "helpers.h"
#include "utils.h"
#include "impl_kalman_filter.cuh"
#include "kalman_filter.h"
#include <chrono>

using namespace std;

void kalman_filter_cpu(
  const Hit* host_hits,
  const Track* host_tracks,
  const std::vector<uint>& event_offsets_hits,
  const std::vector<uint>& event_offsets_tracks,
  State* states_cpu,
  const int max_events) {

  // Event loop
  for ( int event_number = 0; event_number < max_events; ++event_number ) {

    const Hit* host_hits_event = host_hits + event_offsets_hits[event_number];
    const Track* host_tracks_event = host_tracks + event_offsets_tracks[event_number];
    const int number_of_tracks = event_offsets_tracks[event_number+1] - event_offsets_tracks[event_number];

    // Every track will result in one state -> use same offsets for access
    State* states_event = states_cpu + event_offsets_tracks[event_number];

    // Track loop
    for ( int track_number = 0; track_number < number_of_tracks; ++track_number) {
      const MiniState state_at_beamline;
      const Track& track = host_tracks_event[track_number];

      State state = simplified_fit(host_hits_event, state_at_beamline, track);
      //std::cout << "at track " << track_number << ", tx = " << state.tx << ", ty = " << state.ty << ", x = " << state.x << ", y = " << state.y << ", z = " << state.z << std::endl;
      states_event[track_number] = state;
    }

  }
}


extern "C" int run(unsigned const max_events, std::string const input_path,
                   unsigned const n_repetitions, int const device_id,
                   int const)
{
  /* Chose device to use */
  CUDA_ASSERT( hipSetDevice(device_id) );

  std::vector<std::string> folder_contents_hits = list_folder(input_path + "hits", "bin");
  std::vector<std::string> folder_contents_tracks = list_folder(input_path + "tracks", "bin");

  if ( max_events > folder_contents_hits.size() || max_events > folder_contents_tracks.size() ) {
    std::cout << "Requested " << max_events << " events, but only " << folder_contents_hits.size() << " are present in hits directory, and " << folder_contents_tracks.size() << " are present in tracks directory, stopping" << std::endl;
    return 1;
  }

  /* allocate host arrays */
  Hit *host_hits = new Hit[max_hits_per_event * max_events]; // for all hits in all events
  Track *host_tracks = new Track[max_tracks_per_event * max_events]; // for all tracks in all events
  State *states_cpu = new State[max_tracks_per_event * max_events]; // for all states in all events

  /* Also store hits in an SoA format to copy to the GPU */
  HitsSoA host_hits_soa;
  host_hits_soa.x = new my_float_t[max_hits_per_event * max_events];
  host_hits_soa.y = new my_float_t[max_hits_per_event * max_events];
  host_hits_soa.z = new my_float_t[max_hits_per_event * max_events];
  host_hits_soa.mcp = new unsigned int[max_hits_per_event * max_events];

  std::vector<uint> event_offsets_hits;
  get_hits(host_hits, host_hits_soa, folder_contents_hits, event_offsets_hits, input_path, max_events);

  std::vector<uint> event_offsets_tracks;
  get_tracks(host_tracks, folder_contents_tracks, event_offsets_tracks, input_path, max_events);

  /* Run Kalman filter on the CPU */
  kalman_filter_cpu(host_hits, host_tracks, event_offsets_hits, event_offsets_tracks, states_cpu, max_events);


  /* allocate device arrays */
  Track *dev_tracks;
  State *dev_states;
  uint* dev_event_offsets_hits;
  uint* dev_event_offsets_tracks;
  HitsSoA *dev_hits_soa; // Structure holding the pointers to arrays on the device
  HitsSoA *dev_hits_soa_host = new HitsSoA; // Strucutre holding the pointers to arrays on the host

  CUDA_ASSERT( hipMalloc( (void**)&dev_tracks, max_tracks_per_event * max_events * sizeof(Track) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_states, max_tracks_per_event * max_events * sizeof(State) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_event_offsets_hits, (max_events + 1) * sizeof(uint) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_event_offsets_tracks, (max_events + 1) * sizeof(uint) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_hits_soa, sizeof(HitsSoA) ) );
  CUDA_ASSERT( hipMalloc( (void**)&(dev_hits_soa_host->x), max_hits_per_event * max_events * sizeof(my_float_t) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_hits_soa_host->y, max_hits_per_event * max_events * sizeof(my_float_t) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_hits_soa_host->z, max_hits_per_event * max_events * sizeof(my_float_t) ) );
  CUDA_ASSERT( hipMalloc( (void**)&dev_hits_soa_host->mcp, max_hits_per_event * max_events * sizeof(unsigned int) ) );

  State *states_gpu = new State[max_tracks_per_event * max_events];
  const uint total_number_of_hits = event_offsets_hits.back();
  const uint total_number_of_tracks = event_offsets_tracks.back();

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  for ( int repetition = 0; repetition < n_repetitions; ++repetition) {
    /* Copy hits, tracks and offsets from host to device */
    CUDA_ASSERT( hipMemcpy( dev_tracks, host_tracks, total_number_of_tracks * sizeof(Track), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_event_offsets_hits, event_offsets_hits.data(), (max_events + 1) * sizeof(uint), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_event_offsets_tracks, event_offsets_tracks.data(), (max_events + 1) * sizeof(uint), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_hits_soa_host->x, host_hits_soa.x, total_number_of_hits * sizeof(my_float_t), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_hits_soa_host->y, host_hits_soa.y, total_number_of_hits * sizeof(my_float_t), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_hits_soa_host->z, host_hits_soa.z, total_number_of_hits * sizeof(my_float_t), hipMemcpyHostToDevice ) );
    CUDA_ASSERT( hipMemcpy( dev_hits_soa_host->mcp, host_hits_soa.mcp, total_number_of_hits * sizeof(unsigned int), hipMemcpyHostToDevice ) );
    /* We have to copy the pointers to the arrays from the host to the device to be able to only pass the struct and not every array separately */
    CUDA_ASSERT( hipMemcpy( dev_hits_soa, dev_hits_soa_host, sizeof(HitsSoA), hipMemcpyHostToDevice ) );

    dim3 blocks(max_events);
    dim3 threads(32);
    kalman_filter_gpu<<<blocks, threads>>>(dev_hits_soa, dev_tracks, dev_event_offsets_hits, dev_event_offsets_tracks, dev_states);

    /* Copy back states to host */
    CUDA_ASSERT( hipMemcpy( states_gpu, dev_states, total_number_of_tracks * sizeof(State), hipMemcpyDeviceToHost ) );

    hipDeviceSynchronize();
  }

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;

  /* Compare states computed on CPU and on GPU */
  compare_results(states_cpu, states_gpu, total_number_of_tracks);

  cout << "Total duration: " << elapsed_seconds.count() << " s " << endl;
  cout << "Time per event: " << elapsed_seconds.count() / max_events << endl;

  /* Free host arrays */
  delete [] host_hits;
  delete [] host_tracks;
  delete [] states_cpu;
  delete [] states_gpu;
  delete [] host_hits_soa.x;
  delete [] host_hits_soa.y;
  delete [] host_hits_soa.z;
  delete [] host_hits_soa.mcp;

  /* Free device arrays */
  CUDA_ASSERT( hipFree( dev_tracks ) );
  CUDA_ASSERT( hipFree( dev_states ) );
  CUDA_ASSERT( hipFree( dev_event_offsets_hits ) );
  CUDA_ASSERT( hipFree( dev_event_offsets_tracks ) );
  CUDA_ASSERT( hipFree( dev_hits_soa_host->x ) );
  CUDA_ASSERT( hipFree( dev_hits_soa_host->y ) );
  CUDA_ASSERT( hipFree( dev_hits_soa_host->z ) );
  CUDA_ASSERT( hipFree( dev_hits_soa_host->mcp ) );
  CUDA_ASSERT( hipFree( dev_hits_soa ) );

  return 0;
}
