/**
   Run Kalman filter on hits belonging to tracks of LHCb's Velo detector

   author: Dorothea vom Bruch (dorothea.vom.bruch@cern.ch)
   date: 05/2019

 */

#include "helpers.h"
#include "utils.h"
#include "impl_kalman_filter.cuh"
#include "kalman_filter.h"
#include <chrono>

using namespace std;

void kalman_filter_cpu(
  const Hit* host_hits,
  const Track* host_tracks,
  const std::vector<uint>& event_offsets_hits,
  const std::vector<uint>& event_offsets_tracks,
  State* states_cpu,
  const int max_events) {
  
  // Event loop
  for ( int event_number = 0; event_number < max_events; ++event_number ) {
    
    const Hit* host_hits_event = host_hits + event_offsets_hits[event_number];
    const Track* host_tracks_event = host_tracks + event_offsets_tracks[event_number];
    const int number_of_tracks = event_offsets_tracks[event_number+1] - event_offsets_tracks[event_number];
        
    // Every track will result in one state -> use same offsets for access
    State* states_event = states_cpu + event_offsets_tracks[event_number];
    
    // Track loop
    for ( int track_number = 0; track_number < number_of_tracks; ++track_number) {
      const MiniState state_at_beamline;
      const Track& track = host_tracks_event[track_number];
      
      State state = simplified_fit(host_hits_event, state_at_beamline, track);
      //std::cout << "at track " << track_number << ", tx = " << state.tx << ", ty = " << state.ty << ", x = " << state.x << ", y = " << state.y << ", z = " << state.z << std::endl;
      states_event[track_number] = state;
    }
    
  }
}

extern "C" int run(unsigned const max_events, std::string const input_path,
                   unsigned const n_repetitions, int const device_id,
                   int const n_streams)
{
    
  /* Chose device to use */
  CUDA_ASSERT( hipSetDevice(device_id) );
  
  std::vector<std::string> folder_contents_hits = list_folder(input_path + "hits", "bin");
  std::vector<std::string> folder_contents_tracks = list_folder(input_path + "tracks", "bin");
   
  if ( max_events > folder_contents_hits.size() || max_events > folder_contents_tracks.size() ) {
    std::cout << "Requested " << max_events << " events, but only " << folder_contents_hits.size() << " are present in hits directory, and " << folder_contents_tracks.size() << " are present in tracks directory, stopping" << std::endl;
    return 1;
  }
  
  /* allocate host arrays */
  Hit *host_hits = new Hit[max_hits_per_event * max_events]; // for all hits in all events 
  Track *host_tracks = new Track[max_tracks_per_event * max_events]; // for all tracks in all events
  State *states_cpu = new State[max_tracks_per_event * max_events]; // for all states in all events
  State *states_gpu[n_streams];

   for (int i_stream = 0; i_stream < n_streams; i_stream++) {
     states_gpu[i_stream] = new State[max_tracks_per_event * max_events]; // for all states in all events
   }
  
  std::vector<uint> event_offsets_hits;
  get_hits(host_hits, folder_contents_hits, event_offsets_hits, input_path, max_events);
     
  std::vector<uint> event_offsets_tracks;
  get_tracks(host_tracks, folder_contents_tracks, event_offsets_tracks, input_path, max_events);

  /* Run Kalman filter on the CPU */
  kalman_filter_cpu(host_hits, host_tracks, event_offsets_hits, event_offsets_tracks, states_cpu, max_events);
  
  /* allocate device arrays */
  Hit *dev_hits[n_streams];
  Track *dev_tracks[n_streams];
  State *dev_states[n_streams];
  uint* dev_event_offsets_hits[n_streams];
  uint* dev_event_offsets_tracks[n_streams];

  for (int i_stream = 0; i_stream < n_streams; i_stream++) {
    CUDA_ASSERT( hipMalloc( (void**)&dev_hits[i_stream], n_streams * max_hits_per_event * max_events * sizeof(Hit) ) );
    CUDA_ASSERT( hipMalloc( (void**)&dev_tracks[i_stream], n_streams * max_tracks_per_event * max_events * sizeof(Track) ) );
    CUDA_ASSERT( hipMalloc( (void**)&dev_states[i_stream], n_streams * max_tracks_per_event * max_events * sizeof(State) ) );
    CUDA_ASSERT( hipMalloc( (void**)&dev_event_offsets_hits[i_stream], n_streams * (max_events + 1) * sizeof(uint) ) );
    CUDA_ASSERT( hipMalloc( (void**)&dev_event_offsets_tracks[i_stream], n_streams * (max_events + 1) * sizeof(uint) ) );
  }

  /* Create streams */
  hipStream_t *stream = new hipStream_t[n_streams];
  for (int i_stream = 0; i_stream < n_streams; i_stream++) {
    hipStreamCreate(&stream[i_stream]);
  }
  
  /* Copy hits, tracks and offsets from host to device */
  const uint total_number_of_hits = event_offsets_hits.back();
  const uint total_number_of_tracks = event_offsets_tracks.back();

  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();

  for ( int repitition = 0; repitition < n_repetitions; ++repitition) {
    for (int i_stream = 0; i_stream < n_streams; i_stream++) {
      CUDA_ASSERT( hipMemcpyAsync( dev_hits[i_stream], host_hits, total_number_of_hits * sizeof(Hit), hipMemcpyHostToDevice, stream[i_stream] ) );
      
      CUDA_ASSERT( hipMemcpyAsync( dev_tracks[i_stream], host_tracks, total_number_of_tracks * sizeof(Track), hipMemcpyHostToDevice, stream[i_stream] ) );
      CUDA_ASSERT( hipMemcpyAsync( dev_event_offsets_hits[i_stream], event_offsets_hits.data(), (max_events + 1) * sizeof(uint), hipMemcpyHostToDevice, stream[i_stream] ) );
      CUDA_ASSERT( hipMemcpyAsync( dev_event_offsets_tracks[i_stream], event_offsets_tracks.data(), (max_events + 1) * sizeof(uint), hipMemcpyHostToDevice, stream[i_stream] ) );
    }
    
    /* Set grid dimensions */
    dim3 blocks(max_events);
    dim3 threads(32);
    
    /* Launch kernels */
    for (int i_stream = 0; i_stream < n_streams; i_stream++) {
      kalman_filter_gpu<<<blocks, threads, 0, stream[i_stream]>>>(dev_hits[i_stream], dev_tracks[i_stream], dev_event_offsets_hits[i_stream], dev_event_offsets_tracks[i_stream], dev_states[i_stream]);
    }
    
    /* Copy back states to host */
    for (int i_stream = 0; i_stream < n_streams; i_stream++) {
      CUDA_ASSERT( hipMemcpyAsync( states_gpu[i_stream], dev_states[i_stream], total_number_of_tracks * sizeof(State), hipMemcpyDeviceToHost, stream[i_stream] ) );
    }

    for (int i_stream = 0; i_stream < n_streams; i_stream++) {
      hipStreamSynchronize(stream[i_stream]);
    }
  }
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  
  /* Compare states computed on CPU and on GPU */
  for (int i_stream = 0; i_stream < n_streams; i_stream++) {
    compare_results(states_cpu, states_gpu[i_stream], total_number_of_tracks);
  }

  cout << "Total duration: " << elapsed_seconds.count() << " s " << endl;
  cout << "Time per event: " << elapsed_seconds.count() / max_events / n_streams / n_repetitions << endl;
  
  /* Free host arrays */
  delete [] host_hits;
  delete [] host_tracks;
  delete [] states_cpu;
 for (int i_stream = 0; i_stream < n_streams; i_stream++) {
   delete [] states_gpu[i_stream];
 }
 
  /* Free device arrays */
  for (int i_stream = 0; i_stream < n_streams; i_stream++) {
    CUDA_ASSERT( hipFree( dev_hits[i_stream] ) );
    CUDA_ASSERT( hipFree( dev_tracks[i_stream] ) );
    CUDA_ASSERT( hipFree( dev_states[i_stream] ) );
    CUDA_ASSERT( hipFree( dev_event_offsets_hits[i_stream] ) );
    CUDA_ASSERT( hipFree( dev_event_offsets_tracks[i_stream] ) );
  }

  /* Destroy streams */
  for (int i_stream = 0; i_stream < n_streams; i_stream++) {
    CUDA_ASSERT(hipStreamDestroy(stream[i_stream]) );
  }
  
  return 0;
}
